#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#include<stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <sys/time.h>

__global__ void find_prime(int N,int* a,float* b,int* c)
{	
	//*p_size = s+1;
	//__shared__ cuda_count = 1;
	//__shared__ cuda_largest = 2;
	
	int i = blockIdx.x;
	//atomicAdd(&cuda_count,i);
	//__syncthreads();
	//to find a[i] is prime or not
	int flag = 0;

	for(int j=3;j<=b[i];j=j+2)
	{
		if(a[i]%j==0)
		{
			flag = 1;	
			c[i] = -1;
			break;	
		}
	}
}

int* prime_numbers;
float* sqrt_prime;
int *cuda_prime;
float *cuda_sqrt;
int *is_prime;
int *cuda_is_prime;
int main(int argc,char *args[])
{
	if(argc!=3)
	{
		printf("./GPU_Prime -t Problem_Size\n");
		return 0;
	}
	struct timeval time;
	int count = 1;
	int largest = 2;
	int problem_size = atoi(args[2]);
	printf("Problem Size %d\n",problem_size);
	int no_of_elements = 0;
	problem_size = problem_size - 2;//for 1 and 2
	if(problem_size%2 == 0)
	{
		no_of_elements = problem_size/2;
	}
	else
	{
		no_of_elements = problem_size/2 + 1;
	}
	
	prime_numbers = (int *)malloc(no_of_elements*sizeof(int));
	sqrt_prime = (float *)malloc(no_of_elements*sizeof(float));
	is_prime = (int *)malloc(no_of_elements*sizeof(int));	
	
	int h = 3;
	for(int f=0;f<no_of_elements;f++)
	{
		prime_numbers[f] = h;
		sqrt_prime[f] = sqrt(h);
		is_prime[f] = 1;
		//printf("prime[%d] = %d sqrt[%d] = %f\n",f,prime_numbers[f],f,sqrt_prime[f]);
		h = h+2; 
	}
		
	gettimeofday(&time,NULL);
	double t1 = time.tv_sec + (time.tv_usec/1000000.0);

	hipMalloc ( (void**)&cuda_prime, no_of_elements * sizeof (int) );
        hipMalloc ( (void**)&cuda_sqrt, no_of_elements * sizeof (float) );
        hipMalloc ( (void**)&cuda_is_prime, no_of_elements * sizeof (int) );


	
	hipMemcpy( cuda_prime, prime_numbers, no_of_elements * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy( cuda_sqrt, sqrt_prime, no_of_elements * sizeof(float), hipMemcpyHostToDevice);
	//cudaMemcpy( cuda_is_prime, is_prime, no_of_elements * sizeof(int), cudaMemcpyHostToDevice);
	find_prime<<<no_of_elements,1>>>(no_of_elements,cuda_prime,cuda_sqrt,cuda_is_prime);
	
	hipMemcpy( is_prime, cuda_is_prime , no_of_elements * sizeof(int), hipMemcpyDeviceToHost);
	
	for(int g=0;g<no_of_elements;g++)
	{	
		//printf("is_prime[%d] = %d number %d\n",g,is_prime[g],prime_numbers[g]);
		if(is_prime[g]!=-1)
		{
			count++;
			largest = prime_numbers[g];
		}
	}
	printf("Count %d\n Largest %d\n",count,largest);
	gettimeofday(&time,NULL);
	double t2 = time.tv_sec + (time.tv_usec / 1000000.0);
	printf("Time Taken %f \n",t2-t1);
	hipFree(cuda_prime);
	hipFree(cuda_sqrt);
	hipFree(cuda_is_prime);

	
		
	
}
