#include "stdio.h"
#include <math.h>
#include <stdlib.h>
//#include "mpi.h"
#include <sys/time.h>
#include <stdint.h>
#include<hip/hip_runtime.h>
#include<math.h>

#include<string.h>


__global__ void find_prime(int N,int* a,float* b,int* c)
{
        //*p_size = s+1;
        //__shared__ cuda_count = 1;
        //__shared__ cuda_largest = 2;

        int i = blockIdx.x;
        //atomicAdd(&cuda_count,i);
        //__syncthreads();
        //to find a[i] is prime or not
        int flag = 0;

        for(int j=3;j<=b[i];j=j+2)
        {
                if(a[i]%j==0)
                {
                        flag = 1;
                        c[i] = -1;
                        break;
                }
        }
}


extern "C" int* invoke_cuda_find_prime(int from,int to,int r)
{	
	printf("from %d to %d\n",from,to);
	int n = to-from+1;
	if(from ==0)
	{
		from = 3;
		n = to - from+1;
	}
	if(from%2==0)
	{
		from = from+1;
	}
	int* prime_numbers;
	float* sqrt_;
	int* is_prime;
	int index =0;
	int size_of_prime_number=0;
	prime_numbers = (int*)malloc(sizeof(int)*n);
	sqrt_ = (float*)malloc(sizeof(float)*n);
	is_prime = (int*)malloc(sizeof(int)*n);
		
	int* cuda_prime;
	float* cuda_sqrt;
	int*cuda_is_prime;
	int sa=0;
	printf("from %d to %d rank %d\n",from,to,r);

	for(sa=from;sa<=to;sa++)
	{
		//printf("s %d\n",sa,r);
		if(sa%2!=0)
		{
			prime_numbers[index] = sa;
			sqrt_[index] = sqrt(sa);
			is_prime[index] = -1;
			printf("s %d prime[%d] %d sqrt %f rank %d \n",sa,index,prime_numbers[index],sqrt(sa),r);
			index++;
			size_of_prime_number++;
		}
	}
	
	hipMalloc ( (void**)&cuda_prime, size_of_prime_number * sizeof (int) );
        hipMalloc ( (void**)&cuda_sqrt, size_of_prime_number * sizeof (float) );
        hipMalloc ( (void**)&cuda_is_prime, size_of_prime_number * sizeof (int) );
	
	hipMemcpy( cuda_prime, prime_numbers, size_of_prime_number * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy( cuda_sqrt, sqrt_, size_of_prime_number * sizeof(float), hipMemcpyHostToDevice);
        
        find_prime<<<size_of_prime_number,1>>>(size_of_prime_number,cuda_prime,cuda_sqrt,cuda_is_prime);

        hipMemcpy( is_prime, cuda_is_prime , size_of_prime_number * sizeof(int), hipMemcpyDeviceToHost);
	int largest = 0;
	int count = 0;
	for(int g=0;g<size_of_prime_number;g++)
        {
                printf("is_prime[%d] = %d number %d\n",g,is_prime[g],prime_numbers[g]);
                if(is_prime[g]!=-1)
                {	printf("Prime Number %d\n",prime_numbers[g]);
                        count++;
                        largest = prime_numbers[g];
                }
        }
	int* b;
	b = (int*)malloc(sizeof(int)*3);
	b[0] = count;
	b[1] = largest;
        printf("Count %d\n Largest %d\n",count,largest);
        hipFree(cuda_prime);
        hipFree(cuda_sqrt);
	hipFree(cuda_is_prime);
	return b;


}

