#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<string.h>
#include<math.h>
#include <sys/time.h>
#include "rnd.c"
#include "rnd.h"

__global__ void sort(int n_c,float* a,int* b)
{	
	int i = blockIdx.x;
	int len_of_block = b[i];
	for(int j=0;j<len_of_block;j++)
	{
		int k=j;
		while(k>0 && *((float*)a+i*n_c+k) < *((float*)a+i*n_c+k-1))
		{
			float temp = *((float*)a+i*n_c+k);
			*((float*)a+i*n_c+k) = *((float*)a+i*n_c+k-1);
			*((float*)a+i*n_c+k-1) = temp;
			k--;
		}	
	}
	
}

 void display(float* array,int length)
  {
          //printf("size = %d \n",sizeof(array));
          for(int i=0;i<length;i++)
          {
               printf("%f \n",*((float*) array+i));
          }
          printf("\n");
  
  }


void display_(float* array,int length)
{
        //printf("size = %d \n",sizeof(array));
        for(int i=0;i<length;i++)
        {
             printf("%f ",*((float*) array+i));
        }
        printf("\n");

}

float* input;
float* bucket;
float* cuda_bucket;
int* cuda_count;
float* output;

int main(int argc,char *args[])
{
	if(argc!=3)
        {
                printf("./GPU_Prime -t Problem_Size\n");
                return 0;
        }
	int n = atoi(args[2]);
	int max_num = n*10;
	input = (float*)malloc(sizeof(float)*n);
	output = (float*)malloc(sizeof(float)*n);
	random_number_generator_normal(input,n,max_num);
	int no_of_buckets = 4;
	bucket = (float *)malloc(sizeof(float)*no_of_buckets*n);
	int* count;
   	count = (int *)malloc(sizeof(int)*no_of_buckets);
	for(int k=0;k<no_of_buckets;k++)
	{
		count[k] = 0;
	}
	printf("Before :\n");
	display(input,n);
	int bucket_no = 0;
	int limit = max_num/no_of_buckets;
	printf("limit %d\n",limit);
	printf("putting in bucket\n");
	for(int i=0;i<n;i++)
	{
		bucket_no = floor(input[i]/limit);
         	*((float *)bucket+bucket_no*n+count[bucket_no])=input[i] ;
                count[bucket_no] += 1;
		
	}
	for(int j=0;j<no_of_buckets;j++)
	{
		printf("j %d %d ",j,count[j]);
		display_(((float*)bucket+j*n),count[j]);
		
	}
	struct timeval time;
        gettimeofday(&time,NULL);
        double t1 = time.tv_sec + (time.tv_usec/1000000.0);
	
	hipMalloc((void**)&cuda_bucket,sizeof(float)*no_of_buckets*n);
	hipMalloc((void**)&cuda_count,sizeof(int)*no_of_buckets);
	hipMemcpy(cuda_bucket,bucket,sizeof(float)*no_of_buckets*n,hipMemcpyHostToDevice);
	hipMemcpy(cuda_count,count,sizeof(int)*no_of_buckets,hipMemcpyHostToDevice);

	sort<<<no_of_buckets,1>>>(n,cuda_bucket,cuda_count);

	hipMemcpy(bucket,cuda_bucket,sizeof(float)*no_of_buckets*n,hipMemcpyDeviceToHost);
	
	
	printf("Bucket After :\n");
	int o_index = 0;
        for(int j=0;j<no_of_buckets;j++)
        {
                printf("j %d %d ",j,count[j]);
                display_(((float*)bucket+j*n),count[j]);
	        for(int k=0;k<count[j];k++)
                {
                        output[o_index] = *((float*)bucket+j*n+k);
                        o_index++;
                }

        }	
	printf("After :\n");
        display(output,n);


	gettimeofday(&time,NULL);
        double t2 = time.tv_sec + (time.tv_usec/1000000.0);
        printf("Time Taken %f \n",t2-t1);
	//hipFree(cuda_input);
	//hipFree(cuda_answer);

}
